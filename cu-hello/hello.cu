#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "hello.h"

// #define N 10000000 // 1meg
#define N 100000000 // 10meg
#define MAX_ERR 1e-6

void printStatus() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        printf("Device %d:\n", i);
        printf("  Name: %s\n", deviceProp.name);
        printf("  Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Total Global Memory: %zu bytes\n", deviceProp.totalGlobalMem);
        printf("  Multiprocessors: %d\n", deviceProp.multiProcessorCount);

        size_t freeMem, totalMem;
        hipMemGetInfo(&freeMem, &totalMem);
        printf("  Used Memory: %zu bytes\n", totalMem - freeMem);
        printf("  Free Memory: %zu bytes\n", freeMem);
        printf("  Total Memory: %zu bytes\n", totalMem);
        printf("\n");
    }
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    printf("Hello World from GPU!\n");

    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

float *a, *b, *out;
float *d_a, *d_b, *d_out;

int sayHello() {
    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void **)&d_a, sizeof(float) * N);
    hipMalloc((void **)&d_b, sizeof(float) * N);
    hipMalloc((void **)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    vector_add<<<1, 1>>>(d_out, d_a, d_b, N);

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for (int i = 0; i < N; i++) {
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    printf("GPU Status:\n");
    printStatus();

    return 0;
}

int freeMem() {
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a);
    free(b);
    free(out);

    printf("GPU Status:\n");
    printStatus();

    return 0;
}
