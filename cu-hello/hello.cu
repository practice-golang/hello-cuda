#include "hip/hip_runtime.h"
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>

#include "hello.h"

// #define N 10000000 // 1meg
#define N 100000000 // 10meg
#define MAX_ERR 1e-6

void printStatus() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    for (int i = 0; i < deviceCount; ++i) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, i);

        printf("Device %d:\n", i);
        printf("  Name: %s\n", deviceProp.name);
        printf("  Compute Capability: %d.%d\n", deviceProp.major, deviceProp.minor);
        printf("  Total Global Memory: %zu bytes\n", deviceProp.totalGlobalMem);
        printf("  Multiprocessors: %d\n", deviceProp.multiProcessorCount);

        size_t freeMem, totalMem;
        hipMemGetInfo(&freeMem, &totalMem);
        printf("  Used Memory: %zu bytes\n", totalMem - freeMem);
        printf("  Free Memory: %zu bytes\n", freeMem);
        printf("  Total Memory: %zu bytes\n", totalMem);
        printf("\n");
    }
}

__global__ void vector_add(float *out, float *a, float *b, int n) {
    printf("Hello World from GPU!\n");

    for (int i = 0; i < n; i++) {
        out[i] = a[i] + b[i];
    }
}

int *hostData;
int *deviceData;

float *a, *b, *out;
float *d_a, *d_b, *d_out;

int sayHello() {
    // CUDA malloc host memory
    // size_t memSize = N * sizeof(int);
    size_t memSize = 1024 * sizeof(int);
    hipError_t error = hipHostMalloc((void**)&hostData, memSize);
    if (error != hipSuccess) {
        printf("hipHostMalloc returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }

    // CUDA malloc device memory
    error = hipMalloc((void**)&deviceData, memSize);
    if (error != hipSuccess) {
        printf("hipMalloc returned error code %d, line(%d)\n", error, __LINE__);
        exit(EXIT_FAILURE);
    }


    a = (float *)malloc(sizeof(float) * N);
    b = (float *)malloc(sizeof(float) * N);
    out = (float *)malloc(sizeof(float) * N);

    // Initialize array
    for (int i = 0; i < N; i++) {
        a[i] = 1.0f;
        b[i] = 2.0f;
    }

    // Allocate device memory
    hipMalloc((void **)&d_a, sizeof(float) * N);
    hipMalloc((void **)&d_b, sizeof(float) * N);
    hipMalloc((void **)&d_out, sizeof(float) * N);

    // Transfer data from host to device memory
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);

    // Main function
    vector_add<<<1, 1>>>(d_out, d_a, d_b, N);

    // Transfer data back to host memory
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // Verification
    for (int i = 0; i < N; i++) {
        assert(fabs(out[i] - a[i] - b[i]) < MAX_ERR);
    }
    printf("out[0] = %f\n", out[0]);
    printf("PASSED\n");

    printf("GPU Status:\n");
    printStatus();

    return 0;
}

int freeMem() {
    // Deallocate device memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);

    // Deallocate host memory
    free(a);
    free(b);
    free(out);

    // CUDA freee host memory
    hipHostFree(hostData);
    hipFree(deviceData);

    printf("GPU Status:\n");
    printStatus();

    return 0;
}
